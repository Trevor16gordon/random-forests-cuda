#include "hip/hip_runtime.h"
"""
__global__ calculate_gina_scores(float* impurity_sscores,float* X_train,float* y_train,const int unique_classes,const int row,const int dim){
    int Dim = threadIdx.x+blockIdx.x*blockDim.x;
    int Row = threadIdx.y+blockIdx.y*blockDim.y;
    if(Dim < w && Row < l){
        float split_value =X_train[Row * w+ Dim];

        int group1_counts[20] = {0};//Max of 20 dimensions which can be increased
        group2_counts =group1_counts;
        int length1=0;
        int length2=0;
        int sum1=0;
        int sum2=0;

        for(int i=0;i<l;i++){
            if(X_train[i* w+ Dim]>=split_value){
                //Belongs to group 1
                group1_counts[y[i]]++;
                length1++;
            }
            else{
                //Belongs to group 2
                group2_counts[y[i]]++;
                length2++;
            }
        }
        int p1 = length1/(length1+length2);
        int p2 = length2/(length1+length2);

        if(length1 > 0){
            for(int i=0;i<unique_classes;i++){
                sum1+=(group1_counts*group1_counts)/(length1*length1);
            }
        }
        if(length2 > 0){
            for(int i=0;i<unique_classes;i++){
                sum2+=(group2_counts*group2_counts)/(length2*length2);
            }
        }

        impurity = p1*sum1+p2*sum2;
        // Write our new pixel value out
        impurity_scores[Row * w + Dim] = (impurity);

    }
}



//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
#define BLOCKSIZE 1024
//send an array of indices called index(start from 0 to l-1) which is the 2nd argument in here from python itself.
__global__ find_best_gina_score(float* index,float* all_gina_scores, const int l,const int w){
    //loading segment of data in local memory
	__shared__ float scan_array[2*BLOCKSIZE];
	unsigned int t =threadIdx.x;
	unsigned int start=2*blockIdx.x*blockDim.x;

	if(start+t <len){
		scan_array[t]=all_gina_scores[start+t];
	}
    else{
        scan_array[t]=0;
    }

	if(start+blockDim.x+t <len){
		scan_array[blockDim.x+t]=all_gina_scores[start+blockDim.x+t];
	}
	else{
		scan_array[blockDim.x+t]=0;
	}

    for (unsigned int stride = blockDim.x;stride > 0; stride /= 2){
        __syncthreads();
        if (t < stride){
           if(partialSum[t] < partialSum[t+stride]){
               partialSum[t]=partialSum[t+stride];
               index[t]=index[t+stride];
           }
        }            
    }
    //This returns max value and index at the 1st index i.e. 0 in all_gina_scores and index matrices respectively
}
"""